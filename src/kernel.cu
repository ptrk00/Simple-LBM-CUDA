#include "hip/hip_runtime.h"
#include ""
#include "automata.h"

static constexpr unsigned int BLOCK_SIZE = 10;

int LBM_Kernel() {

    dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 grid(NX / block.x, NY / block.y, 1);


    collision << <grid, block >> > ();

    hipDeviceSynchronize();

    streaming << <grid, block >> > ();

    hipDeviceSynchronize();

    return 0;
}

